#include <iostream>
#include "hip/hip_runtime.h"
#include <hipblas.h>
#define BLOCK_DIM 32
#define TILE_DIM 32
#define BLOCK_ROWS 8


template<class T>
class MatrixHost {
    T *values;
    long matrix_size;
public:
    explicit MatrixHost(long n) : matrix_size(n) {
        values = new T[matrix_size * matrix_size * sizeof(T)];
    }

    void Initialize_Matrix(void) {
        time_t t;
        srand((unsigned) time(&t));
        for (int j = 0; j < matrix_size * matrix_size; j++) {
            values[j] = (T) (rand() & 0xFF) / 10.0f;
        }
    }

    T *get_values(void) {
        return values;
    }

    long get_size() {
        return sizeof(T) * matrix_size * matrix_size;
    }

    ~MatrixHost() {
        delete[]values;
    }
};


template<class T>
class MatrixDevice {
    T *values;
    long matrix_size;
public:
    explicit MatrixDevice(long n) : matrix_size(n) {
        hipMalloc((T **) &values, matrix_size * matrix_size * sizeof(T));
    };

    T *get_values(void) {
        return values;
    }

    ~MatrixDevice() {
        hipFree(values);
    }
};

template<typename type>
__global__ void IJK(const type *A, const type *B, type *C, int matrix_size) {
    int i = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int j = blockIdx.y * BLOCK_DIM + threadIdx.y;
    float temp = 0;
    for (int k = 0; k < matrix_size; k++) {
        temp += A[i * matrix_size + k] * B[k * matrix_size + j];
    }
    C[i * matrix_size + j] = temp;
}

template<typename type>
__global__ void IKJ(const type *A, const type *B, type *C, int matrix_size) {
    int i = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int k = blockIdx.y * BLOCK_DIM + threadIdx.y;
    float reg = A[i * matrix_size + k];
    for (int j = 0; j < matrix_size; j++) {
        atomicAdd(&C[i * matrix_size + j], reg * B[k * matrix_size + j]);
    }
}


template<typename type>
__global__ void KIJ(const type *A, const type *B, type *C, int matrix_size) {
    int k = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int i = blockIdx.y * BLOCK_DIM + threadIdx.y;
    float reg = A[i * matrix_size + k];
    for (int j = 0; j < matrix_size; j++) {
        atomicAdd(&C[i * matrix_size + j], reg * B[k * matrix_size + j]);
    }
}


template<typename type>
__global__ void JIK(const type *A, const type *B, type *C, int matrix_size) {
    int j = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int i = blockIdx.y * BLOCK_DIM + threadIdx.y;
    float temp = 0;
    for (int k = 0; k < matrix_size; k++) {
        temp += A[i * matrix_size + k] * B[k * matrix_size + j];
    }
    C[i * matrix_size + j] = temp;
}


template<typename type>
__global__ void JKI(const type *A, const type *B, type *C, int matrix_size) {
    int j = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int k = blockIdx.y * BLOCK_DIM + threadIdx.y;
    float reg = B[k * matrix_size + j];
    for (int i = 0; i < matrix_size; i++) {
        atomicAdd(&C[i * matrix_size + j], A[i * matrix_size + k] * reg);
    }
}

template<typename type>
__global__ void KJI(const type *A, const type *B, type *C, int matrix_size) {
    int k = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int j = blockIdx.y * BLOCK_DIM + threadIdx.y;
    float reg = B[k * matrix_size + j];
    for (int i = 0; i < matrix_size; i++) {
        atomicAdd(&C[i * matrix_size + j], A[i * matrix_size + k] * reg);
    }

}


template<typename type>
void multiply_on_host(const type *A, const type *B, type *C, long matrix_size) { //JIK mult
    for (int j = 0; j < matrix_size; j++) {
        for (int i = 0; i < matrix_size; i++) {
            type r = 0;
            for (int k = 0; k < matrix_size; k++) {
                r += A[(i*matrix_size) + k] * B[(k * matrix_size) + j];
            }
            C[(i* matrix_size) + j] = r;
        }
    }
}


template<typename type>
void checkResult(type *host_array, type *device_array, long matrix_size) {
    double epsilon = 0.5;
    int pos = 0;
    for (long i = 0; i < matrix_size; i++) {
        if (abs(device_array[i] - host_array[i]) > epsilon) {
            std::cout << "ERROR " << i<< "  " << host_array[i] <<" "<<device_array[i];
            return;
        }
    }
    std::cout << "SUCCESS\n";
}


template<typename type>
void cublas_multiply(const type *A, const type *B, type *C, long matrix_size) {
    //int dima = matrix_size,dimB =matrix_size , dimC =matrix_size;
    hipEvent_t start_multiply, stop_multiply;
    float time_multiply;
    hipEventCreate(&start_multiply);
    hipEventCreate(&stop_multiply);
    float alpha = 1.0;
    float betta = 0.0;
    float *alph_point = &alpha;
    float *betta_point = &betta;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipEventRecord(start_multiply);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size, matrix_size, matrix_size, alph_point, A, matrix_size, B,
                matrix_size, betta_point, C, matrix_size);
    hipEventRecord(stop_multiply);
    hipEventSynchronize(stop_multiply);
    hipEventElapsedTime(&time_multiply, start_multiply, stop_multiply);
    hipblasDestroy(handle);
    std::cout << "GPU PERFORMANCE: " << (matrix_size * matrix_size * matrix_size * 1000) / ((double) time_multiply)
              << "FLOPS\n";
}


template<typename type>
__global__ void transposeNaive(type *odata, const type *idata) {
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        odata[x * width + (y + j)] = idata[(y + j) * width + x];
    }
}


int main(int argc, char **argv) {

    long matrix_size = atol(argv[1]);
    MatrixHost<float> A_h(matrix_size);
    MatrixHost<float> B_h(matrix_size);
    MatrixHost<float> C_h(matrix_size);
    MatrixHost<float> C_from_cublas(matrix_size);


    A_h.Initialize_Matrix();
    B_h.Initialize_Matrix();

    //multiply_on_host<float>(A_h.get_values(), B_h.get_values(), C_h.get_values(), matrix_size);

    MatrixDevice<float> A_d(matrix_size);
    MatrixDevice<float> B_d(matrix_size);
    MatrixDevice<float> C_d(matrix_size);
    MatrixDevice<float> A_dt(matrix_size);


    hipMemcpy(A_d.get_values(), A_h.get_values(), A_h.get_size(), hipMemcpyHostToDevice);
    hipMemcpy(B_d.get_values(), B_h.get_values(), A_h.get_size(), hipMemcpyHostToDevice);

    dim3 block(BLOCK_DIM, BLOCK_DIM);
    dim3 grid((matrix_size + block.x - 1) / block.x, (matrix_size + block.y - 1) / block.y);

    //transposeNaive<float> <<<grid,block>>>(A_dt.get_values(),A_d.get_values());

    cublas_multiply<float>(A_d.get_values(), B_d.get_values(), C_d.get_values(), matrix_size);
    //cudaEventRecord(start_device);
    cublas_multiply<float>(A_d.get_values(), B_d.get_values(), C_d.get_values(), matrix_size);
    //cudaEventRecord(stop_device);
    //cudaEventSynchronize(stop_device);
    //cudaEventElapsedTime(&time_device,start_device,stop_device);


    hipMemcpy(C_from_cublas.get_values(), C_d.get_values(), A_h.get_size(), hipMemcpyDeviceToHost);


    //checkResult<float>(C_h.get_values(), C_from_cublas.get_values(), matrix_size * matrix_size);
}