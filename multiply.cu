#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#define BLOCK_DIM 32
#define TILE_DIM 32
#define BLOCK_ROWS 8


template<class T>
class MatrixHost {
    T *values;
    long matrix_size;
public:
    explicit MatrixHost(long n) : matrix_size(n) {
        values = new T[matrix_size * matrix_size* sizeof(T)];
    }
    void Initialize_Matrix(void){
        time_t t;
        srand((unsigned) time(&t));
        for (int j = 0; j < matrix_size*matrix_size; j++) {
            values[j] = (T) (rand() & 0xFF) / 10.0f;
        }
    }

    T * get_values(void) {
        return values;
    }

    long get_size() {
        return sizeof(T) * matrix_size * matrix_size;
    }

    ~MatrixHost() {
        delete []values;
    }
};


template<class T>
class MatrixDevice{
    T *values;
    long matrix_size;
public:
    explicit MatrixDevice(long n) : matrix_size(n) {
        hipMalloc((T **) &values, matrix_size* matrix_size * sizeof(T));
    };

    T *get_values(void) {
        return values;
    }

    ~MatrixDevice() {
        hipFree(values);
    }
};

template <typename type>
__global__ void IJK(const type *A, const type *B,type *C,int matrix_size) {
    int i = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int j = blockIdx.y * BLOCK_DIM + threadIdx.y;
    float temp = 0;
    for (int k = 0; k < matrix_size ; k++) {
        temp += A[i*matrix_size + k] * B[k*matrix_size+ j];
    }
    C[i*matrix_size+ j] = temp;
}

template <typename type>
__global__ void IKJ(const type *A, const type *B,type *C,int matrix_size) {
    int i = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int k = blockIdx.y * BLOCK_DIM + threadIdx.y;
    float reg = A[i*matrix_size+ k];
    for (int j = 0;  j < matrix_size; j++) {
        atomicAdd(&C[i*matrix_size+j],reg * B[k*matrix_size +j]);
    }
}


template <typename type>
__global__ void KIJ(const type *A, const type *B,type *C,int matrix_size) {
    int k = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int i = blockIdx.y * BLOCK_DIM + threadIdx.y;
    float reg = A[i * matrix_size + k];
    for (int j = 0; j < matrix_size; j++) {
        atomicAdd(&C[i*matrix_size + j] , reg * B[k*matrix_size +j]);
    }
}


template <typename type>
__global__ void JIK(const type *A, const type *B,type *C,int matrix_size) {
    int j = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int i = blockIdx.y * BLOCK_DIM + threadIdx.y;
    float temp = 0;
    for (int k = 0; k < matrix_size ; k++) {
        temp += A[i*matrix_size + k] * B[k*matrix_size + j];
    }
    C[i*matrix_size + j] = temp;
}


template <typename type>
__global__ void JKI(const type *A, const type *B,type *C,int matrix_size) {
    int j = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int k = blockIdx.y * BLOCK_DIM + threadIdx.y;
    float reg = B[k*matrix_size + j];
    for (int i = 0; i < matrix_size ; i++) {
        atomicAdd(&C[i*matrix_size+j],A[i*matrix_size + k] * reg);
    }
}

template <typename type>
__global__ void KJI(const type *A, const type *B,type *C,int matrix_size) {
    int k = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int j = blockIdx.y * BLOCK_DIM + threadIdx.y;
    float reg = B[k*matrix_size + j];
    for (int i = 0; i < matrix_size ; i++) {
        atomicAdd(&C[i*matrix_size+j],A[i*matrix_size + k] * reg);
    }

}


template <typename type>
void multiply_on_host(const type *A, const type *B,type *C,int matrix_size) {
    for (int i = 0; i < matrix_size; i++) {
        for (int j = 0; j < matrix_size; j++) {
            float r = 0;
            for (int k = 0; k < matrix_size; k++) {
                r += A[i*matrix_size + k] * B[k*matrix_size+ j];
            }
            C[i*matrix_size+ j] = r;
        }
    }
}
template <typename type>
void checkResult(type *host_array, type *device_array, const long matrix_size) {
    double epsilon = 0.1;
    int pos = 0;
    for (long i = 0; i < matrix_size; i++) {
        if (abs(device_array[i] - host_array[i]) > epsilon) {
            pos++;
        }
    }
    std::cout << "SUCCESS\n"<<pos;
}


template <typename type>
__global__ void cublas_multiply(const type *A, const type *B,type *C, int matrix_size) {
    //int dima = matrix_size,dimB =matrix_size , dimC =matrix_size;
    float alpha = 1.0;
    float betta =0.0;
    float* alph_point= &alpha;
    float* betta_point = &betta;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,matrix_size,matrix_size,matrix_size,alph_point,A,matrix_size,B,matrix_size,betta_point,C,matrix_size);
    hipblasDestroy(handle);
}

template <typename type>
__global__ void transposeNaive(type *odata, const type *idata)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS) {
        odata[x * width + (y + j)] = idata[(y + j) * width + x];
    }
    __syncthreads();
}




int main(int argc,char** argv) {

    hipEvent_t start_host,stop_host,start_device,stop_device;
    float time_host,time_device;
    int version;
    hipEventCreate(&start_host);
    hipEventCreate(&stop_host);
    hipEventCreate(&start_device);
    hipEventCreate(&stop_device);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    cublasGetVersion(handle,&version);
    std::cout<<"cuBLAS version: " << version<<"\n";



    int matrix_size =  atoi(argv[1]);
    MatrixHost<float> A_h (matrix_size);
    MatrixHost<float> B_h (matrix_size);
    MatrixHost<float> C_h(matrix_size);
    MatrixHost<float>C_from_cublas (matrix_size);



    A_h.Initialize_Matrix();
    B_h.Initialize_Matrix();



    hipEventRecord(start_host);
    multiply_on_host<float>(A_h.get_values(),B_h.get_values(),C_h.get_values(),matrix_size);
    hipEventRecord(stop_host);
    hipEventSynchronize(stop_host);
    hipEventElapsedTime(&time_host,start_host,stop_host);


    MatrixDevice <float> A_d(matrix_size);
    MatrixDevice <float> B_d(matrix_size);
    MatrixDevice <float> C_d(matrix_size);
    MatrixDevice <float> A_dt(matrix_size);

    hipMemcpy(A_d.get_values(), A_h.get_values(), A_h.get_size(), hipMemcpyHostToDevice);
    hipMemcpy(B_d.get_values(), B_h.get_values(), A_h.get_size(), hipMemcpyHostToDevice);

    dim3 block(BLOCK_DIM,BLOCK_DIM,1);
    dim3 grid ((matrix_size+block.x-1)/block.x,(matrix_size+block.y-1)/block.y,1 );


    transposeNaive<float> <<<grid,block>>>(A_dt.get_values(),A_d.get_values());
    
    cublas_multiply<float><<<grid,block>>>(A_dt.get_values(),B_d.get_values(),C_d.get_values(),matrix_size);
    hipEventRecord(start_device);
    cublas_multiply<float><<<grid,block>>>(A_dt.get_values(),B_d.get_values(),C_d.get_values(),matrix_size);
    hipEventRecord(stop_device);
    hipEventSynchronize(stop_device);



    hipEventElapsedTime(&time_device,start_device,stop_device);

    hipMemcpy(C_from_cublas.get_values(), C_d.get_values(), A_h.get_size(), hipMemcpyDeviceToHost);




    checkResult<float>(C_h.get_values(),C_from_cublas.get_values(),matrix_size*matrix_size);

    hipblasDestroy(handle);
    std::cout<<"GPU PERFORMANCE: " << (matrix_size*matrix_size*matrix_size)/(time_device)<<" FLOPS\n";
}